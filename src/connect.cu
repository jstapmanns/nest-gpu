#include "hip/hip_runtime.h"
#include <time.h>
#include <sys/time.h>
#include <iostream>
#include <algorithm>
#include <cstdlib>
#include <climits>
#include <vector>
#include <utility>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipcub/hipcub.hpp>
#include "cuda_error.h"
#include "copass_kernels.h"
#include "copass_sort.h"
#include "distribution.h"
#include "connect.h"
#include "nestgpu.h"
#include "utilities.h"

extern __constant__ float NESTGPUTimeResolution;

uint h_MaxNodeNBits;
__device__ uint MaxNodeNBits;
// maximum number of bits used to represent node index 

uint h_MaxPortNBits;
__device__ uint MaxPortNBits;
// maximum number of bits used to represent receptor port index and delays 

uint h_PortMask;
__device__ uint PortMask;
// bit mask used to extract port index

uint *d_ConnGroupNum;
__device__ uint *ConnGroupNum;
// ConnGroupNum[i_spike_buffer]
// Number of connection groups outgoing from node i_spike_buffer
// where i_spike_buffer is the source node index
// Output connections from the source nodes are organized in groups
// All connection of a group have the same delay

uint *d_ConnGroupIdx0;
__device__ uint *ConnGroupIdx0;
// ig0 = ConnGroupIdx0[i_spike_buffer] is the index in the whole
// connection-group array of the first connection group outgoing
// from the node i_spike_buffer

int64_t *d_ConnGroupIConn0;
__device__ int64_t *ConnGroupIConn0;
// i_conn0 = ConnGroupIConn0[ig] with ig = 0, ..., Ng
//  is the index in the whole connection array of the first connection
// belonging to the connection group ig

int64_t *d_ConnGroupNConn;
__device__ int64_t *ConnGroupNConn;
// ConnGroupNConn[ig] with ig = 0, ..., Ng
// Ng: total number of connection groups for the whole network
// number of output connections in the connection group ig
// of the node i_spike_buffer

uint *d_ConnGroupDelay;
__device__ uint *ConnGroupDelay;
// ConnGroupDelay[ig]
// delay associated to all connections of the connection group ig
// with ig = 0, ..., Ng

int64_t NConn; // total number of connections in the whole network

int64_t h_ConnBlockSize = 10000000; // 160000000; //50000000;
__device__ int64_t ConnBlockSize;
// size (i.e. number of connections) of connection blocks 

uint h_MaxDelayNum;

std::vector<uint*> KeySubarray;
__device__ uint** SourceDelayArray;
// Array of source node indexes and delays of all connections
// Source node indexes and delays are merged in a single integer variable
// The most significant MaxNodeNBits are used for the node index 
// the others (less significant) bits are used to represent the delay
// This array is used as a key array for sorting the connections
// in ascending order according to the source node index
// Connections from the same source node are sorted according to
// the delay

std::vector<connection_struct*> ConnectionSubarray;
__device__ connection_struct** ConnectionArray;
// array of target node indexes, receptor port index, synapse type,
// weight of all connections
// used as a value for key-value sorting of the connections (see above)


enum ConnectionFloatParamIndexes {
  i_weight_param = 0,
  i_delay_param,
  N_CONN_FLOAT_PARAM
};

enum ConnectionIntParamIndexes {
  i_source_param = 0,
  i_target_param,
  i_port_param,
  i_syn_group_param,
  N_CONN_INT_PARAM
};

const std::string ConnectionFloatParamName[N_CONN_FLOAT_PARAM] = {
  "weight",
  "delay"
};

const std::string ConnectionIntParamName[N_CONN_INT_PARAM] = {
  "source",
  "target",
  "port",
  "syn_group"
};

__global__ void OrganizeConnectionGroups(uint *key_subarray,
					 uint *key_subarray_prev,
					 int64_t n_block_conn,
					 uint *conn_group_num_tmp,
					 int64_t block_conn_idx0,
					 uint *conn_group_idx0,
					 int64_t *conn_group_iconn0,
					 uint *conn_group_key)
{
  int64_t i_conn = threadIdx.x + blockIdx.x * blockDim.x;
  if (i_conn>=n_block_conn) return;
  uint val = key_subarray[i_conn];
  uint i_neuron = val >> MaxPortNBits;
  int64_t prev_val;
  if (i_conn==0) {
    if (key_subarray_prev != NULL) {
      prev_val = *key_subarray_prev;
    }
    else {
      prev_val = -1;      // just to ensure it is different from val
    }
  }
  else {
    prev_val = key_subarray[i_conn-1];
  }
  if (val != prev_val) {
    uint i_source_conn_group = atomicAdd(&conn_group_num_tmp[i_neuron], 1);
    uint ig0 = conn_group_idx0[i_neuron];
    uint conn_group_idx = ig0 + i_source_conn_group;
    conn_group_iconn0[conn_group_idx] = block_conn_idx0 + i_conn;
    conn_group_key[conn_group_idx] = val;
  }
}


__global__ void checkConnGroups(uint n_neuron, int64_t *source_conn_idx0,
				int64_t *source_conn_num, uint **key_subarray,
				int64_t block_size, int64_t *conn_group_iconn0,
				uint *conn_group_nconn, uint *conn_group_num,
				uint *conn_group_idx0)
{
  const uint i_neuron = threadIdx.x + blockIdx.x * blockDim.x;
  if (i_neuron>=n_neuron) return;
  
  const int64_t nc =source_conn_num[i_neuron];
  const int64_t ic0 = source_conn_idx0[i_neuron];
  const uint ig0 = conn_group_idx0[i_neuron];
  
  int i_source_conn_group = 0;
  for (int64_t ic=ic0; ic<ic0+nc; ic++) {
    uint ib =(uint)(ic / block_size);
    int64_t jc = ic % block_size;
    uint val = key_subarray[ib][jc];
    
    uint prev_val = 0;
    if (jc==0 && ib!=0) {
      prev_val = key_subarray[ib-1][block_size-1];
    }
    else if (jc>0) {
      prev_val = key_subarray[ib][jc-1];
    }
    if (i_source_conn_group==0 || val!=prev_val) {
      uint conn_group_idx = ig0 + i_source_conn_group;
      conn_group_iconn0[conn_group_idx] = ic;
      if (ic > ic0) {
	conn_group_nconn[conn_group_idx - 1] = ic
	  - conn_group_iconn0[conn_group_idx - 1];
      }
      i_source_conn_group++;
    }
  }
  uint conn_group_idx = ig0 + i_source_conn_group;
  conn_group_nconn[conn_group_idx - 1] = ic0 + nc
    - conn_group_iconn0[conn_group_idx - 1];

}

__global__ void getSourceConnNum(uint n_neuron, int64_t *source_conn_idx0,
				 int64_t *source_conn_num, int64_t n_conn)
{
  uint i_neuron = threadIdx.x + blockIdx.x * blockDim.x;
  if (i_neuron>=n_neuron) {
    return;
  }
  if ( i_neuron==(n_neuron-1) ) {
    source_conn_num[i_neuron] = n_conn - source_conn_idx0[i_neuron];
  }
  else {
    source_conn_num[i_neuron] = source_conn_idx0[i_neuron + 1]
      - source_conn_idx0[i_neuron];
  }
}
  

__global__ void countConnectionGroups(uint *key_subarray,
					uint *key_subarray_prev,
					int64_t n_block_conn,
					uint *conn_group_num,
					int64_t block_conn_idx0)
{
  int64_t i_conn = threadIdx.x + blockIdx.x * blockDim.x;
  if (i_conn>=n_block_conn) return;
  uint val = key_subarray[i_conn];
  uint i_neuron = val >> MaxPortNBits;
  int64_t prev_val;
  if (i_conn==0) {
    if (key_subarray_prev != NULL) {
      prev_val = *key_subarray_prev;
    }
    else {
      prev_val = -1;      // just to ensure it is different from val
    }
  }
  else {
    prev_val = key_subarray[i_conn-1];
  }
  if (val != prev_val) {
    atomicAdd(&conn_group_num[i_neuron], 1);
  }
}


bool print_sort_err = true;
bool print_sort_cfr = false;
bool compare_with_serial = false;
uint last_i_sub = 0;

__global__ void setWeights(connection_struct *conn_subarray, float weight,
			   int64_t n_conn)
{
  int64_t i_conn = threadIdx.x + blockIdx.x * blockDim.x;
  if (i_conn>=n_conn) return;
  conn_subarray[i_conn].weight = weight;
}

__global__ void setWeights(connection_struct *conn_subarray, float *arr_val,
			   int64_t n_conn)
{
  int64_t i_conn = threadIdx.x + blockIdx.x * blockDim.x;
  if (i_conn>=n_conn) return;
  conn_subarray[i_conn].weight = arr_val[i_conn];
}

__global__ void setDelays(uint *key_subarray, float *arr_val,
			  int64_t n_conn, float time_resolution)
{
  int64_t i_conn = threadIdx.x + blockIdx.x * blockDim.x;
  if (i_conn>=n_conn) return;
  int delay = (int)round(arr_val[i_conn]/time_resolution);
  delay = max(delay,1);
  key_subarray[i_conn] = (key_subarray[i_conn] << MaxPortNBits) | delay;
}

__global__ void setDelays(uint *key_subarray, float fdelay,
			  int64_t n_conn, float time_resolution)
{
  int64_t i_conn = threadIdx.x + blockIdx.x * blockDim.x;
  if (i_conn>=n_conn) return;
  int delay = (int)round(fdelay/time_resolution);
  delay = max(delay,1);
  key_subarray[i_conn] = (key_subarray[i_conn] << MaxPortNBits) | delay;
}

__global__ void setPort(connection_struct *conn_subarray, uint port,
			   int64_t n_conn)
{
  int64_t i_conn = threadIdx.x + blockIdx.x * blockDim.x;
  if (i_conn>=n_conn) return;
  conn_subarray[i_conn].target_port =
    (conn_subarray[i_conn].target_port << MaxPortNBits) | port; 
}

__global__ void setSynGroup(connection_struct *conn_subarray,
			    unsigned char syn_group,
			    int64_t n_conn)
{
  int64_t i_conn = threadIdx.x + blockIdx.x * blockDim.x;
  if (i_conn>=n_conn) return;
  conn_subarray[i_conn].syn_group = syn_group; 
}

__global__ void getConnGroupNConn(int64_t *conn_group_iconn0,
				  int64_t *conn_group_nconn,
				  uint conn_group_num, int64_t n_conn)
{
  uint conn_group_idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (conn_group_idx >= conn_group_num) return;
  else if (conn_group_idx == (conn_group_num - 1)) {
    conn_group_nconn[conn_group_num - 1] = n_conn
      - conn_group_iconn0[conn_group_num - 1];
  }
  else {
    conn_group_nconn[conn_group_idx] = conn_group_iconn0[conn_group_idx + 1]
      - conn_group_iconn0[conn_group_idx];
  }
}

__global__ void getConnGroupDelay(uint *conn_group_key,
				  uint *conn_group_delay,
				  uint conn_group_num)
{
  uint conn_group_idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (conn_group_idx >= conn_group_num) return;
  conn_group_delay[conn_group_idx] = conn_group_key[conn_group_idx]
    & PortMask;
}

int allocateNewBlocks(std::vector<uint*> &key_subarray,
		      std::vector<connection_struct*> &conn_subarray,
		      int64_t block_size, uint new_n_block)
{
  // Allocating GPU memory for new connection blocks
  // allocate new blocks if needed
  for (uint ib=key_subarray.size(); ib<new_n_block; ib++) {
    uint *d_key_pt;
    connection_struct *d_connection_pt;
    // allocate GPU memory for new blocks 
    gpuErrchk(hipMalloc(&d_key_pt, block_size*sizeof(uint)));
    gpuErrchk(hipMalloc(&d_connection_pt,
			 block_size*sizeof(connection_struct)));
    key_subarray.push_back(d_key_pt);
    conn_subarray.push_back(d_connection_pt);
  }

  return 0;
}


int setConnectionWeights(hiprandGenerator_t &gen, void *d_storage,
			 connection_struct *conn_subarray, int64_t n_conn,
			 SynSpec &syn_spec)
{
  if (syn_spec.weight_distr_ >= DISTR_TYPE_ARRAY   // probability distribution
      && syn_spec.weight_distr_ < N_DISTR_TYPE) {  // or array
    if (syn_spec.weight_distr_ == DISTR_TYPE_ARRAY) {
      gpuErrchk(hipMemcpy(d_storage, syn_spec.weight_h_array_pt_,
			   n_conn*sizeof(float), hipMemcpyHostToDevice));    
    }
    else if (syn_spec.weight_distr_ == DISTR_TYPE_NORMAL_CLIPPED) {
      CURAND_CALL(hiprandGenerateUniform(gen, (float*)d_storage, n_conn));
      randomNormalClipped((float*)d_storage, n_conn, syn_spec.weight_mu_,
			  syn_spec.weight_sigma_, syn_spec.weight_low_,
			  syn_spec.weight_high_);
    }
    else if (syn_spec.weight_distr_==DISTR_TYPE_NORMAL) {
      float low = syn_spec.weight_mu_ - 5.0*syn_spec.weight_sigma_;
      float high = syn_spec.weight_mu_ + 5.0*syn_spec.weight_sigma_;
      CURAND_CALL(hiprandGenerateUniform(gen, (float*)d_storage, n_conn));
      randomNormalClipped((float*)d_storage, n_conn, syn_spec.weight_mu_,
			  syn_spec.weight_sigma_, low, high);
    }
    else {
      throw ngpu_exception("Invalid connection weight distribution type");
    }
    setWeights<<<(n_conn+1023)/1024, 1024>>>
      (conn_subarray, (float*)d_storage, n_conn);
    DBGCUDASYNC
  }
  else {
    setWeights<<<(n_conn+1023)/1024, 1024>>>
      (conn_subarray, syn_spec.weight_, n_conn);
    DBGCUDASYNC
  }
    
  return 0;
}


int setConnectionDelays(hiprandGenerator_t &gen, void *d_storage,
			uint *key_subarray, int64_t n_conn,
			SynSpec &syn_spec, float time_resolution)
{
  if (syn_spec.delay_distr_ >= DISTR_TYPE_ARRAY   // probability distribution
      && syn_spec.delay_distr_ < N_DISTR_TYPE) {  // or array
    if (syn_spec.delay_distr_ == DISTR_TYPE_ARRAY) {
      gpuErrchk(hipMemcpy(d_storage, syn_spec.delay_h_array_pt_,
			   n_conn*sizeof(float), hipMemcpyHostToDevice));
    }
    else if (syn_spec.delay_distr_ == DISTR_TYPE_NORMAL_CLIPPED) {
      CURAND_CALL(hiprandGenerateUniform(gen, (float*)d_storage, n_conn));
      randomNormalClipped((float*)d_storage, n_conn, syn_spec.delay_mu_,
			  syn_spec.delay_sigma_, syn_spec.delay_low_,
			  syn_spec.delay_high_);
    }
    else if (syn_spec.delay_distr_ == DISTR_TYPE_NORMAL) {
      float low = syn_spec.delay_mu_ - 5.0*syn_spec.delay_sigma_;
      float high = syn_spec.delay_mu_ + 5.0*syn_spec.delay_sigma_;
      CURAND_CALL(hiprandGenerateUniform(gen, (float*)d_storage, n_conn));
      randomNormalClipped((float*)d_storage, n_conn, syn_spec.delay_mu_,
			  syn_spec.delay_sigma_, syn_spec.delay_low_,
			  syn_spec.delay_high_);
    }
    else {
      throw ngpu_exception("Invalid connection delay distribution type");
    }

    setDelays<<<(n_conn+1023)/1024, 1024>>>
      (key_subarray, (float*)d_storage, n_conn, time_resolution);
    DBGCUDASYNC

  }
  else {
    setDelays<<<(n_conn+1023)/1024, 1024>>>
      (key_subarray, syn_spec.delay_, n_conn, time_resolution);
    DBGCUDASYNC
  }
  return 0;
}


int organizeConnections(float time_resolution, uint n_node, int64_t n_conn,
			int64_t block_size,
			std::vector<uint*> &key_subarray,
			std::vector<connection_struct*> &conn_subarray)
{
  typedef uint key_t;
  timeval startTV;
  timeval endTV;
  hipDeviceSynchronize();
  gettimeofday(&startTV, NULL);

  if (n_conn > 0) {
    printf("Allocating auxiliary GPU memory...\n");
    int64_t storage_bytes = 0;
    void *d_storage = NULL;
    copass_sort::sort<uint, connection_struct>(key_subarray.data(),
					       conn_subarray.data(),
					       n_conn, block_size, d_storage,
					       storage_bytes);
    printf("storage bytes: %ld\n", storage_bytes);
    gpuErrchk(hipMalloc(&d_storage, storage_bytes));
    
    printf("Sorting...\n");
    copass_sort::sort<uint, connection_struct>(key_subarray.data(),
					       conn_subarray.data(), n_conn,
					       block_size, d_storage,
					       storage_bytes);
  
    // free temporarily allocated storage
    gpuErrchk(hipFree(d_storage));
    storage_bytes = 0; 
  }
  printf("Indexing connection groups...\n");
  uint k = key_subarray.size();

  gpuErrchk(hipMalloc(&d_ConnGroupNum, n_node*sizeof(uint)));
  gpuErrchk(hipMemset(d_ConnGroupNum, 0, n_node*sizeof(uint)));
  
  uint *key_subarray_prev = NULL;
  for (uint i=0; i<k; i++) {
    uint n_block_conn = i<(k-1) ? block_size : n_conn - block_size*(k-1);
    countConnectionGroups<<<(n_block_conn+1023)/1024, 1024>>>
      (key_subarray[i], key_subarray_prev, n_block_conn, d_ConnGroupNum,
       block_size*i);
    DBGCUDASYNC
      
    key_subarray_prev = key_subarray[i] + block_size - 1;
  }
  
  gpuErrchk(hipMalloc(&d_ConnGroupIdx0, (n_node+1)*sizeof(uint)));  
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;

  // Determine temporary device storage requirements for prefix sum
  hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
				d_ConnGroupNum, d_ConnGroupIdx0,
				n_node+1);
  // Allocate temporary storage for prefix sum
  gpuErrchk(hipMalloc(&d_temp_storage, temp_storage_bytes));

  // Run exclusive prefix sum
  hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
				d_ConnGroupNum, d_ConnGroupIdx0,
				n_node+1);
  gpuErrchk(hipFree(d_temp_storage));  // free temporary allocated storage

  uint tot_conn_group_num;
  gpuErrchk(hipMemcpy(&tot_conn_group_num, &d_ConnGroupIdx0[n_node],
		       sizeof(uint), hipMemcpyDeviceToHost));
  printf("Total number of connection groups: %d\n", tot_conn_group_num);
  
  
  //////////////////////////////////////////////////////////////////////
  if (tot_conn_group_num > 0) {
    int64_t *d_conn_group_iconn0_unsorted;
    gpuErrchk(hipMalloc(&d_conn_group_iconn0_unsorted,
			 tot_conn_group_num*sizeof(int64_t)));
  
    uint *d_conn_group_key_unsorted;
    gpuErrchk(hipMalloc(&d_conn_group_key_unsorted,
			 tot_conn_group_num*sizeof(uint)));
  
    gpuErrchk(hipMemset(d_ConnGroupNum, 0, n_node*sizeof(uint)));
    key_subarray_prev = NULL;
    for (uint i=0; i<k; i++) {
      uint n_block_conn = i<(k-1) ? block_size : n_conn - block_size*(k-1);
      OrganizeConnectionGroups<<<(n_block_conn+1023)/1024, 1024>>>
	(key_subarray[i], key_subarray_prev, n_block_conn,
	 d_ConnGroupNum, block_size*i, d_ConnGroupIdx0,
	 d_conn_group_iconn0_unsorted, d_conn_group_key_unsorted);
      DBGCUDASYNC
	key_subarray_prev = key_subarray[i] + block_size - 1;
    }

    gpuErrchk(hipMalloc(&d_ConnGroupIConn0,
			 tot_conn_group_num*sizeof(int64_t)));
    uint *d_conn_group_key;
    gpuErrchk(hipMalloc(&d_conn_group_key,
			 tot_conn_group_num*sizeof(uint)));
    void *d_conn_group_storage = NULL;
    size_t conn_group_storage_bytes = 0;

    // Determine temporary storage requirements for sorting connection groups
    hipcub::DeviceRadixSort::SortPairs(d_conn_group_storage,
				    conn_group_storage_bytes,
				    d_conn_group_key_unsorted,
				    d_conn_group_key,
				    d_conn_group_iconn0_unsorted,
				    d_ConnGroupIConn0,
				    tot_conn_group_num);
    // Allocate temporary storage for sorting
    gpuErrchk(hipMalloc(&d_conn_group_storage, conn_group_storage_bytes));
    // Run radix sort
    hipcub::DeviceRadixSort::SortPairs(d_conn_group_storage,
				    conn_group_storage_bytes,
				    d_conn_group_key_unsorted,
				    d_conn_group_key,
				    d_conn_group_iconn0_unsorted,
				    d_ConnGroupIConn0,
				    tot_conn_group_num);
    gpuErrchk(hipFree(d_conn_group_storage));
    gpuErrchk(hipFree(d_conn_group_iconn0_unsorted));
    gpuErrchk(hipFree(d_conn_group_key_unsorted));

    gpuErrchk(hipMalloc(&d_ConnGroupNConn,
			 tot_conn_group_num*sizeof(int64_t)));
  
    getConnGroupNConn<<<(tot_conn_group_num+1023)/1024, 1024>>>
      (d_ConnGroupIConn0, d_ConnGroupNConn, tot_conn_group_num, n_conn);
    DBGCUDASYNC
    gpuErrchk(hipMalloc(&d_ConnGroupDelay,
			 tot_conn_group_num*sizeof(uint)));
  
    getConnGroupDelay<<<(tot_conn_group_num+1023)/1024, 1024>>>
      (d_conn_group_key, d_ConnGroupDelay, tot_conn_group_num);
    DBGCUDASYNC
    
    gpuErrchk(hipFree(d_conn_group_key));
  }
  // find maxumum number of connection groups (delays) over all neurons
  uint *d_max_delay_num = NULL;  
  d_temp_storage = NULL;
  temp_storage_bytes = 0;  
  // Determine temporary device storage requirements
  hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes,
			 d_ConnGroupNum, d_max_delay_num, n_node);
  // Allocate temporary storage
  gpuErrchk(hipMalloc(&d_temp_storage, temp_storage_bytes));
  gpuErrchk(hipMalloc(&d_max_delay_num, sizeof(uint)));
  
  // Run maximum search
  hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes,
			 d_ConnGroupNum, d_max_delay_num, n_node);
	    
  CUDASYNC
  gpuErrchk(hipFree(d_temp_storage)); // free temporary allocated storage  

  gpuErrchk(hipMemcpy(&h_MaxDelayNum, d_max_delay_num,
		       sizeof(uint), hipMemcpyDeviceToHost));
  gpuErrchk(hipFree(d_max_delay_num));

  printf("Maximum number of connection groups (delays) over all nodes: %d\n",
	 h_MaxDelayNum);

  gettimeofday(&endTV, NULL);
  long time = (long)((endTV.tv_sec * 1000000.0 + endTV.tv_usec)
		     - (startTV.tv_sec * 1000000.0 + startTV.tv_usec));
  printf("%-40s%.2f ms\n", "Time: ", (double)time / 1000.);
  printf("Done\n");
  
  
  return 0;
}



__global__ void NewConnectInitKernel(uint *conn_group_idx0,
				     uint *conn_group_num,
				     int64_t *conn_group_iconn0,
				     int64_t *conn_group_nconn,
				     uint *conn_group_delay,
				     int64_t block_size,
				     uint **source_delay_array,
				     connection_struct **connection_array)
{
  
  ConnGroupIdx0 = conn_group_idx0;
  ConnGroupNum = conn_group_num;
  ConnGroupIConn0 = conn_group_iconn0;
  ConnGroupNConn = conn_group_nconn;
  ConnGroupDelay = conn_group_delay;
  ConnBlockSize = block_size;
  SourceDelayArray = source_delay_array;
  ConnectionArray = connection_array;
}

int NewConnectInit()
{
  uint k = ConnectionSubarray.size();
  uint **d_source_delay_array;
  gpuErrchk(hipMalloc(&d_source_delay_array, k*sizeof(uint*)));
  gpuErrchk(hipMemcpy(d_source_delay_array, KeySubarray.data(),
		       k*sizeof(uint*), hipMemcpyHostToDevice));
  
  connection_struct **d_connection_array;
  gpuErrchk(hipMalloc(&d_connection_array, k*sizeof(connection_struct*)));
  gpuErrchk(hipMemcpy(d_connection_array, ConnectionSubarray.data(),
		       k*sizeof(connection_struct*), hipMemcpyHostToDevice));

  NewConnectInitKernel<<<1,1>>>(d_ConnGroupIdx0, d_ConnGroupNum,
				d_ConnGroupIConn0, d_ConnGroupNConn,
				d_ConnGroupDelay, h_ConnBlockSize,
				d_source_delay_array,
				d_connection_array);
  DBGCUDASYNC

  return 0;
}

__global__ void setMaxNodeNBitsKernel(int max_node_nbits, int max_port_nbits,
				      int port_mask)
{
  MaxNodeNBits = max_node_nbits;
  MaxPortNBits = max_port_nbits;
  PortMask = port_mask;
}

int setMaxNodeNBits(int max_node_nbits)
{
  h_MaxNodeNBits = max_node_nbits;
  h_MaxPortNBits = 32 - h_MaxNodeNBits;
  h_PortMask = (1 << h_MaxPortNBits) - 1;
  setMaxNodeNBitsKernel<<<1,1>>>(h_MaxNodeNBits, h_MaxPortNBits, h_PortMask);
  DBGCUDASYNC

  return 0;
}  

int *sortArray(int *h_arr, int n_elem)
{
  // allocate unsorted and sorted array in device memory
  int *d_arr_unsorted;
  int *d_arr_sorted;
  gpuErrchk(hipMalloc(&d_arr_unsorted, n_elem*sizeof(int)));
  gpuErrchk(hipMalloc(&d_arr_sorted, n_elem*sizeof(int)));
  gpuErrchk(hipMemcpy(d_arr_unsorted, h_arr, n_elem*sizeof(int),
		       hipMemcpyHostToDevice));
  void *d_storage = NULL;
  size_t storage_bytes = 0;
  // Determine temporary storage requirements for sorting source indexes
  hipcub::DeviceRadixSort::SortKeys(d_storage, storage_bytes, d_arr_unsorted,
				 d_arr_sorted, n_elem);
  // Allocate temporary storage for sorting
  gpuErrchk(hipMalloc(&d_storage, storage_bytes));
  // Run radix sort
  hipcub::DeviceRadixSort::SortKeys(d_storage, storage_bytes, d_arr_unsorted,
				 d_arr_sorted, n_elem);
  gpuErrchk(hipFree(d_storage));
  gpuErrchk(hipFree(d_arr_unsorted));

  return d_arr_sorted;
}

__global__ void setSourceTargetIndexKernel(int64_t n_src_tgt, int  n_source,
					   int n_target, int64_t *d_src_tgt_arr,
					   int *d_src_arr, int *d_tgt_arr)
{
  int64_t i_src_tgt = (int64_t)blockIdx.x * blockDim.x + threadIdx.x; 
  if (i_src_tgt >= n_src_tgt) return;
  int i_src =(int)(i_src_tgt / n_target);
  int i_tgt =(int)(i_src_tgt % n_target);
  int src_id = d_src_arr[i_src];
  int tgt_id = d_tgt_arr[i_tgt];
  int64_t src_tgt_id = ((int64_t)src_id << 32) | tgt_id;
  d_src_tgt_arr[i_src_tgt] = src_tgt_id;
  //printf("i_src_tgt %lld\tsrc_id %d\ttgt_id %d\tsrc_tgt_id %lld\n", 
  //	 i_src_tgt, src_id, tgt_id, src_tgt_id); 
}

// Count number of connections per source-target couple
__global__ void CountConnectionsKernel(int64_t n_conn, int n_source,
				       int n_target, int64_t *src_tgt_arr,
				       int64_t *src_tgt_conn_num,
				       int syn_group)
{
  int64_t i_conn = (int64_t)blockIdx.x * blockDim.x + threadIdx.x; 
  if (i_conn >= n_conn) return;

  uint i_block = (uint)(i_conn / ConnBlockSize);
  int64_t i_block_conn = i_conn % ConnBlockSize;
  connection_struct conn = ConnectionArray[i_block][i_block_conn];
  if (syn_group==-1 || conn.syn_group == syn_group) {
    // First get target node index
    uint target_port = conn.target_port;
    int i_target = target_port >> MaxPortNBits;
    uint source_delay = SourceDelayArray[i_block][i_block_conn];
    int i_source = source_delay >> MaxPortNBits;
    int64_t i_src_tgt = ((int64_t)i_source << 32) | i_target;
    int64_t i_arr = locate(i_src_tgt, src_tgt_arr, n_source*n_target);
    if (src_tgt_arr[i_arr] == i_src_tgt) {
      //printf("i_conn %lld i_source %d i_target %d i_src_tgt %lld "
      //     "i_arr %lld\n", i_conn, i_source, i_target, i_src_tgt, i_arr);
      // (atomic)increase the number of connections for source-target couple
      atomicAdd((unsigned long long *)&src_tgt_conn_num[i_arr], 1);
    }
  }
}


// Fill array of connection indexes
__global__ void SetConnectionsIndexKernel(int64_t n_conn, int n_source,
					  int n_target, int64_t *src_tgt_arr,
					  int64_t *src_tgt_conn_num,
					  int64_t *src_tgt_conn_cumul,
					  int syn_group, int64_t *conn_ids)
{
  int64_t i_conn = (int64_t)blockIdx.x * blockDim.x + threadIdx.x; 
  if (i_conn >= n_conn) return;

  uint i_block = (uint)(i_conn / ConnBlockSize);
  int64_t i_block_conn = i_conn % ConnBlockSize;
  connection_struct conn = ConnectionArray[i_block][i_block_conn];
  if (syn_group==-1 || conn.syn_group == syn_group) {
    // First get target node index
    uint target_port = conn.target_port;
    int i_target = target_port >> MaxPortNBits;
    uint source_delay = SourceDelayArray[i_block][i_block_conn];
    int i_source = source_delay >> MaxPortNBits;
    int64_t i_src_tgt = ((int64_t)i_source << 32) | i_target;
    int64_t i_arr = locate(i_src_tgt, src_tgt_arr, n_source*n_target);
    if (src_tgt_arr[i_arr] == i_src_tgt) {
      //printf("i_conn %lld i_source %d i_target %d i_src_tgt %lld "
      //     "i_arr %lld\n", i_conn, i_source, i_target, i_src_tgt, i_arr);
      // (atomic)increase the number of connections for source-target couple
      int64_t pos =
	atomicAdd((unsigned long long *)&src_tgt_conn_num[i_arr], 1);
      //printf("pos %lld src_tgt_conn_cumul[i_arr] %lld\n",
      //     pos, src_tgt_conn_cumul[i_arr]);
      conn_ids[src_tgt_conn_cumul[i_arr] + pos] = i_conn;
    }
  }
}


int64_t *NESTGPU::GetConnections(int *i_source_pt, int n_source,
				 int *i_target_pt, int n_target,
				 int syn_group, int64_t *n_conn)
{  
  int64_t *h_conn_ids = NULL;
  int64_t *d_conn_ids = NULL;
  int64_t n_src_tgt = (int64_t)n_source * n_target;
  int64_t n_conn_ids = 0;
  
  if (n_src_tgt > 0) {
    // sort source node index array in GPU memory
    int *d_src_arr = sortArray(i_source_pt, n_source);
    // sort target node index array in GPU memory
    int *d_tgt_arr = sortArray(i_target_pt, n_target);
    // Allocate array of combined source-target indexes (src_arr x tgt_arr)
    int64_t *d_src_tgt_arr;
    gpuErrchk(hipMalloc(&d_src_tgt_arr, n_src_tgt*sizeof(int64_t)));
    // Fill it with combined source-target indexes
    setSourceTargetIndexKernel<<<(n_src_tgt+1023)/1024, 1024>>>
      (n_src_tgt, n_source, n_target, d_src_tgt_arr, d_src_arr, d_tgt_arr);
    // Allocate array of number of connections per source-target couple
    // and initialize it to 0
    int64_t *d_src_tgt_conn_num;
    gpuErrchk(hipMalloc(&d_src_tgt_conn_num, (n_src_tgt + 1)*sizeof(int64_t)));
    gpuErrchk(hipMemset(d_src_tgt_conn_num, 0,
			 (n_src_tgt + 1)*sizeof(int64_t)));

    // Count number of connections per source-target couple
    CountConnectionsKernel<<<(NConn+1023)/1024, 1024>>>
      (NConn, n_source, n_target, d_src_tgt_arr, d_src_tgt_conn_num, syn_group);
    // Evaluate exclusive sum of connections per source-target couple
    // Allocate array for cumulative sum
    int64_t *d_src_tgt_conn_cumul;
    gpuErrchk(hipMalloc(&d_src_tgt_conn_cumul,
			 (n_src_tgt + 1)*sizeof(int64_t)));
    // Determine temporary device storage requirements
    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
				  d_src_tgt_conn_num,
				  d_src_tgt_conn_cumul,
				  n_src_tgt + 1);
    // Allocate temporary storage
    gpuErrchk(hipMalloc(&d_temp_storage, temp_storage_bytes));
    // Run exclusive prefix sum
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
				  d_src_tgt_conn_num,
				  d_src_tgt_conn_cumul,
				  n_src_tgt + 1);
    gpuErrchk(hipFree(d_temp_storage));
    
    // The last element is the total number of required connection Ids
    hipMemcpy(&n_conn_ids, &d_src_tgt_conn_cumul[n_src_tgt],
	       sizeof(int64_t), hipMemcpyDeviceToHost);
    
    if (n_conn_ids > 0) {
      // Allocate array of connection indexes
      gpuErrchk(hipMalloc(&d_conn_ids, n_conn_ids*sizeof(int64_t)));  
      // Set number of connections per source-target couple to 0 again
      gpuErrchk(hipMemset(d_src_tgt_conn_num, 0,
			   (n_src_tgt + 1)*sizeof(int64_t)));
      // Fill array of connection indexes
      SetConnectionsIndexKernel<<<(NConn+1023)/1024, 1024>>>
	(NConn, n_source, n_target, d_src_tgt_arr, d_src_tgt_conn_num,
	 d_src_tgt_conn_cumul, syn_group, d_conn_ids);

      /// check if allocating with new is more appropriate
      h_conn_ids = (int64_t*)malloc(n_conn_ids*sizeof(int64_t));
      gpuErrchk(hipMemcpy(h_conn_ids, d_conn_ids,
			   n_conn_ids*sizeof(int64_t),
			   hipMemcpyDeviceToHost));
	
      gpuErrchk(hipFree(d_src_tgt_arr));
      gpuErrchk(hipFree(d_src_tgt_conn_num));
      gpuErrchk(hipFree(d_src_tgt_conn_cumul));
      gpuErrchk(hipFree(d_conn_ids));
    }
  }
  *n_conn = n_conn_ids;
  
  return h_conn_ids;
}

//////////////////////////////////////////////////////////////////////
// CUDA Kernel that gets all parameters of an array of n_conn connections,
// identified by the indexes conn_ids[i], and puts them in the arrays
// i_source, i_target, port, syn_group, delay, weight
//////////////////////////////////////////////////////////////////////
__global__ void GetConnectionStatusKernel
(int64_t *conn_ids, int64_t n_conn, int *i_source, int *i_target,
 int *port, unsigned char *syn_group, float *delay, float *weight)
{
  int64_t i_arr = (int64_t)blockIdx.x * blockDim.x + threadIdx.x; 
  if (i_arr >= n_conn) return;

   // get connection index, connection block index and index within block
  int64_t i_conn = conn_ids[i_arr];
  uint i_block = (uint)(i_conn / ConnBlockSize);
  int64_t i_block_conn = i_conn % ConnBlockSize;
  // get connection structure
  connection_struct conn = ConnectionArray[i_block][i_block_conn];
  // Get joined target-port parameter, then target index and port index
  uint target_port = conn.target_port;
  i_target[i_arr] = target_port >> MaxPortNBits;
  port[i_arr] = target_port & PortMask;
  // Get weight and synapse group
  weight[i_arr] = conn.weight;
  syn_group[i_arr] = conn.syn_group;
  // Get joined source-delay parameter, then source index and delay
  uint source_delay = SourceDelayArray[i_block][i_block_conn];
  i_source[i_arr] = source_delay >> MaxPortNBits;
  int i_delay = source_delay & PortMask;
  delay[i_arr] = NESTGPUTimeResolution * i_delay;
}

//////////////////////////////////////////////////////////////////////
// CUDA Kernel that gets a float parameter of an array of n_conn connections,
// identified by the indexes conn_ids[i], and puts it in the array
// param_arr
//////////////////////////////////////////////////////////////////////
__global__ void GetConnectionFloatParamKernel
(int64_t *conn_ids, int64_t n_conn, float *param_arr, int i_param)
{
  int64_t i_arr = (int64_t)blockIdx.x * blockDim.x + threadIdx.x; 
  if (i_arr >= n_conn) return;

   // get connection index, connection block index and index within block
  int64_t i_conn = conn_ids[i_arr];
  uint i_block = (uint)(i_conn / ConnBlockSize);
  int64_t i_block_conn = i_conn % ConnBlockSize;
  // get connection structure
  connection_struct conn = ConnectionArray[i_block][i_block_conn];
  switch (i_param) {
  case i_weight_param: {
    param_arr[i_arr] = conn.weight;
    break;
  }
  case i_delay_param: {
    // Get joined source-delay parameter, then delay
    uint source_delay = SourceDelayArray[i_block][i_block_conn];
    int i_delay = source_delay & PortMask;
    param_arr[i_arr] = NESTGPUTimeResolution * i_delay;
    break;
  }
  }
}

//////////////////////////////////////////////////////////////////////
// CUDA Kernel that gets an integer parameter of an array of n_conn connections,
// identified by the indexes conn_ids[i], and puts it in the array
// param_arr
//////////////////////////////////////////////////////////////////////
__global__ void GetConnectionIntParamKernel
(int64_t *conn_ids, int64_t n_conn, int *param_arr, int i_param)
{
  int64_t i_arr = (int64_t)blockIdx.x * blockDim.x + threadIdx.x; 
  if (i_arr >= n_conn) return;

   // get connection index, connection block index and index within block
  int64_t i_conn = conn_ids[i_arr];
  uint i_block = (uint)(i_conn / ConnBlockSize);
  int64_t i_block_conn = i_conn % ConnBlockSize;
  // get connection structure
  connection_struct conn = ConnectionArray[i_block][i_block_conn];
  switch (i_param) {
  case i_source_param: {
    // Get joined source-delay parameter, then source index and delay
    uint source_delay = SourceDelayArray[i_block][i_block_conn];
    param_arr[i_arr] = source_delay >> MaxPortNBits;
    break;
  }
  case i_target_param: {
    // Get joined target-port parameter, then target index
    param_arr[i_arr] = conn.target_port >> MaxPortNBits;
    break;
  }
  case i_port_param: {
    // Get joined target-port parameter, then port index
    param_arr[i_arr] = conn.target_port & PortMask;
    break;
  }
  case i_syn_group_param: {
    // Get synapse group
    param_arr[i_arr] = conn.syn_group;
    break;
  }
  }
}

//////////////////////////////////////////////////////////////////////
// CUDA Kernel that sets a float parameter of an array of n_conn connections,
// identified by the indexes conn_ids[i], using values from the array
// param_arr
//////////////////////////////////////////////////////////////////////
__global__ void SetConnectionFloatParamKernel
(int64_t *conn_ids, int64_t n_conn, float *param_arr, int i_param)
{
  int64_t i_arr = (int64_t)blockIdx.x * blockDim.x + threadIdx.x; 
  if (i_arr >= n_conn) return;

   // get connection index, connection block index and index within block
  int64_t i_conn = conn_ids[i_arr];
  uint i_block = (uint)(i_conn / ConnBlockSize);
  int64_t i_block_conn = i_conn % ConnBlockSize;
  // get connection structure
  connection_struct *conn = &ConnectionArray[i_block][i_block_conn];
  switch (i_param) {
  case i_weight_param: {
    conn->weight = param_arr[i_arr]; 
    break;
  }
  }
}

//////////////////////////////////////////////////////////////////////
// CUDA Kernel that sets a float parameter of an array of n_conn connections,
// identified by the indexes conn_ids[i], to the value val
//////////////////////////////////////////////////////////////////////
__global__ void SetConnectionFloatParamKernel
(int64_t *conn_ids, int64_t n_conn, float val, int i_param)
{
  int64_t i_arr = (int64_t)blockIdx.x * blockDim.x + threadIdx.x; 
  if (i_arr >= n_conn) return;

   // get connection index, connection block index and index within block
  int64_t i_conn = conn_ids[i_arr];
  uint i_block = (uint)(i_conn / ConnBlockSize);
  int64_t i_block_conn = i_conn % ConnBlockSize;
  // get connection structure
  connection_struct *conn = &ConnectionArray[i_block][i_block_conn];
  switch (i_param) {
  case i_weight_param: {
    conn->weight = val; 
    break;
  }
  }
}

//////////////////////////////////////////////////////////////////////
// CUDA Kernel that sets an integer parameter of an array of n_conn connections,
// identified by the indexes conn_ids[i], using values from the array
// param_arr
//////////////////////////////////////////////////////////////////////
__global__ void SetConnectionIntParamKernel
(int64_t *conn_ids, int64_t n_conn, int *param_arr, int i_param)
{
  int64_t i_arr = (int64_t)blockIdx.x * blockDim.x + threadIdx.x; 
  if (i_arr >= n_conn) return;

   // get connection index, connection block index and index within block
  int64_t i_conn = conn_ids[i_arr];
  uint i_block = (uint)(i_conn / ConnBlockSize);
  int64_t i_block_conn = i_conn % ConnBlockSize;
  // get connection structure
  connection_struct *conn = &ConnectionArray[i_block][i_block_conn];
  switch (i_param) {
  case i_target_param: {
    // Get port index from joined target-port parameter
    int i_port = conn->target_port & PortMask;
    // Set joined target-port parameter
    conn->target_port = (param_arr[i_arr] << MaxPortNBits) | i_port;
    break;
  }
  case i_port_param: {
    // Get target index from joined target-port parameter
    int i_target = conn->target_port >> MaxPortNBits;
    // Set joined target-port parameter
    conn->target_port = (i_target << MaxPortNBits) | param_arr[i_arr];
    break;
  }
  case i_syn_group_param: {
    // Set synapse group
    conn->syn_group = param_arr[i_arr]; 
    break;
  }
  }
}

//////////////////////////////////////////////////////////////////////
// CUDA Kernel that sets an integer parameter of an array of n_conn connections,
// identified by the indexes conn_ids[i], to the value val
//////////////////////////////////////////////////////////////////////
__global__ void SetConnectionIntParamKernel
(int64_t *conn_ids, int64_t n_conn, int val, int i_param)
{
  int64_t i_arr = (int64_t)blockIdx.x * blockDim.x + threadIdx.x; 
  if (i_arr >= n_conn) return;

   // get connection index, connection block index and index within block
  int64_t i_conn = conn_ids[i_arr];
  uint i_block = (uint)(i_conn / ConnBlockSize);
  int64_t i_block_conn = i_conn % ConnBlockSize;
  // get connection structure
  connection_struct *conn = &ConnectionArray[i_block][i_block_conn];
  switch (i_param) {
  case i_target_param: {
    // Get port index from joined target-port parameter
    int i_port = conn->target_port & PortMask;
    // Set joined target-port parameter
    conn->target_port = (val << MaxPortNBits) | i_port;
    break;
  }
  case i_port_param: {
    // Get target index from joined target-port parameter
    int i_target = conn->target_port >> MaxPortNBits;
    // Set joined target-port parameter
    conn->target_port = (i_target << MaxPortNBits) | val;
    break;
  }
  case i_syn_group_param: {
    // Set synapse group
    conn->syn_group = val; 
    break;
  }
  }
}


//////////////////////////////////////////////////////////////////////
// Get all parameters of an array of n_conn connections, identified by
// the indexes conn_ids[i], and put them in the arrays
// i_source, i_target, port, syn_group, delay, weight
// NOTE: host arrays should be pre-allocated to store n_conn elements
//////////////////////////////////////////////////////////////////////
int NESTGPU::GetConnectionStatus(int64_t *conn_ids, int64_t n_conn,
				 int *i_source, int *i_target, int *port,
				 unsigned char *syn_group, float *delay,
				 float *weight)
{
  if (n_conn > 0) {
    // declare pointers to arrays in device memory
    int64_t *d_conn_ids;
    int *d_source;
    int *d_target;
    int *d_port;
    unsigned char *d_syn_group;
    float *d_delay;
    float *d_weight;

    // allocate array of connection ids in device memory
    // and copy the ids from host to device array
    gpuErrchk(hipMalloc(&d_conn_ids, n_conn*sizeof(int64_t)));
    gpuErrchk(hipMemcpy(d_conn_ids, conn_ids, n_conn*sizeof(int64_t),
			 hipMemcpyHostToDevice));

    // allocate arrays of connection parameters in device memory
    gpuErrchk(hipMalloc(&d_source, n_conn*sizeof(int)));
    gpuErrchk(hipMalloc(&d_target, n_conn*sizeof(int)));
    gpuErrchk(hipMalloc(&d_port, n_conn*sizeof(int)));
    gpuErrchk(hipMalloc(&d_syn_group, n_conn*sizeof(unsigned char)));
    gpuErrchk(hipMalloc(&d_delay, n_conn*sizeof(float)));
    gpuErrchk(hipMalloc(&d_weight, n_conn*sizeof(float)));
    // host arrays
    
    // launch kernel to get connection parameters
    GetConnectionStatusKernel<<<(n_conn+1023)/1024, 1024 >>>
      (d_conn_ids, n_conn, d_source, d_target, d_port, d_syn_group,
       d_delay, d_weight);

    // copy connection parameters from device to host memory
    gpuErrchk(hipMemcpy(i_source, d_source, n_conn*sizeof(int),
			 hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(i_target, d_target, n_conn*sizeof(int),
			 hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(port, d_port, n_conn*sizeof(int),
			 hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(syn_group, d_syn_group,
			 n_conn*sizeof(unsigned char),
			 hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(delay, d_delay, n_conn*sizeof(float),
			 hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(weight, d_weight, n_conn*sizeof(float),
			 hipMemcpyDeviceToHost));
  }
  
  return 0;
}


// Get the index of the connection float parameter param_name
// if param_name is not a float parameter, return -1
int NESTGPU::GetConnectionFloatParamIndex(std::string param_name)
{
  for (int i=0; i<N_CONN_FLOAT_PARAM; i++) {
    if (param_name==ConnectionFloatParamName[i]) {
      return i;
    }
  }
  
  return -1;
}

// Get the index of the connection int parameter param_name
// if param_name is not an int parameter, return -1
int NESTGPU::GetConnectionIntParamIndex(std::string param_name)
{
  for (int i=0; i<N_CONN_INT_PARAM; i++) {
    if (param_name==ConnectionIntParamName[i]) {
      return i;
    }
  }
  
  return -1;
}

// Check if param_name is a connection float parameter
int NESTGPU::IsConnectionFloatParam(std::string param_name)
{
  if (GetConnectionFloatParamIndex(param_name) >=0 ) {
    return 1;
  }
  else {
    return 0;
  }
}

// Check if param_name is a connection integer parameter
int NESTGPU::IsConnectionIntParam(std::string param_name)
{
  if (GetConnectionIntParamIndex(param_name) >=0 ) {
    return 1;
  }
  else {
    return 0;
  }
}

//////////////////////////////////////////////////////////////////////
// Get the float parameter param_name of an array of n_conn connections,
// identified by the indexes conn_ids[i], and put it in the array
// h_param_arr
// NOTE: host array should be pre-allocated to store n_conn elements
//////////////////////////////////////////////////////////////////////
int NESTGPU::GetConnectionFloatParam(int64_t *conn_ids, int64_t n_conn,
				     float *h_param_arr,
				     std::string param_name)
{
  // Check if param_name is a connection float parameter
  int i_param = GetConnectionFloatParamIndex(param_name);
  if (i_param < 0) {
    throw ngpu_exception(std::string("Unrecognized connection float parameter ")
			 + param_name);
  }
  if (n_conn > 0) {
    // declare pointers to arrays in device memory
    int64_t *d_conn_ids;
    float *d_arr;
    // allocate array of connection ids in device memory
    // and copy the ids from host to device array
    gpuErrchk(hipMalloc(&d_conn_ids, n_conn*sizeof(int64_t)));
    gpuErrchk(hipMemcpy(d_conn_ids, conn_ids, n_conn*sizeof(int64_t),
			 hipMemcpyHostToDevice));
    
    // allocate connection parameter array in device memory
    gpuErrchk(hipMalloc(&d_arr, n_conn*sizeof(float)));
    
    // launch kernel to get connection parameters
    GetConnectionFloatParamKernel<<<(n_conn+1023)/1024, 1024 >>>
      (d_conn_ids, n_conn, d_arr, i_param);
    
    // copy connection parameter array from device to host memory
    gpuErrchk(hipMemcpy(h_param_arr, d_arr, n_conn*sizeof(float),
			 hipMemcpyDeviceToHost));
    // free allocated device memory
    gpuErrchk(hipFree(d_conn_ids));
    gpuErrchk(hipFree(d_arr));
  }
  
  return 0;
}

//////////////////////////////////////////////////////////////////////
// Get the integer parameter param_name of an array of n_conn connections,
// identified by the indexes conn_ids[i], and put it in the array
// h_param_arr
// NOTE: host array should be pre-allocated to store n_conn elements
//////////////////////////////////////////////////////////////////////
int NESTGPU::GetConnectionIntParam(int64_t *conn_ids, int64_t n_conn,
				   int *h_param_arr,
				   std::string param_name)
{
  // Check if param_name is a connection integer parameter
  int i_param = GetConnectionIntParamIndex(param_name);
  if (i_param < 0) {
    throw ngpu_exception(std::string("Unrecognized connection "
				     "integer parameter ") + param_name);
  }
  if (n_conn > 0) {
    // declare pointers to arrays in device memory
    int64_t *d_conn_ids;
    int *d_arr;
    // allocate array of connection ids in device memory
    // and copy the ids from host to device array
    gpuErrchk(hipMalloc(&d_conn_ids, n_conn*sizeof(int64_t)));
    gpuErrchk(hipMemcpy(d_conn_ids, conn_ids, n_conn*sizeof(int64_t),
			 hipMemcpyHostToDevice));
    
    // allocate connection parameter array in device memory
    gpuErrchk(hipMalloc(&d_arr, n_conn*sizeof(int)));
    
    // launch kernel to get connection parameters
    GetConnectionIntParamKernel<<<(n_conn+1023)/1024, 1024 >>>
      (d_conn_ids, n_conn, d_arr, i_param);
    
    // copy connection parameter array from device to host memory
    gpuErrchk(hipMemcpy(h_param_arr, d_arr, n_conn*sizeof(int),
			 hipMemcpyDeviceToHost));
    // free allocated device memory
    gpuErrchk(hipFree(d_conn_ids));
    gpuErrchk(hipFree(d_arr));
  }
  
  return 0;
}


//////////////////////////////////////////////////////////////////////
// Set the float parameter param_name of an array of n_conn connections,
// identified by the indexes conn_ids[i], using values from a distribution
// or from an array
//////////////////////////////////////////////////////////////////////
int NESTGPU::SetConnectionFloatParamDistr(int64_t *conn_ids, int64_t n_conn,
					  std::string param_name)
{
  // Check if param_name is a connection float parameter
  int i_param = GetConnectionFloatParamIndex(param_name);
  if (i_param < 0) {
    throw ngpu_exception(std::string("Unrecognized connection float parameter ")
			 + param_name);
  }
  if (i_param == i_delay_param) {
    throw ngpu_exception("Connection delay cannot be modified");
  }

  if (n_conn > 0) {
    // declare pointers to arrays in device memory
    int64_t *d_conn_ids;
    // allocate array of connection ids in device memory
    // and copy the ids from host to device array
    gpuErrchk(hipMalloc(&d_conn_ids, n_conn*sizeof(int64_t)));
    gpuErrchk(hipMemcpy(d_conn_ids, conn_ids, n_conn*sizeof(int64_t),
			 hipMemcpyHostToDevice));
    
    // get values from array or distribution
    float *d_arr = distribution_->getArray(n_conn);
    
    // launch kernel to set connection parameters
    SetConnectionFloatParamKernel<<<(n_conn+1023)/1024, 1024 >>>
      (d_conn_ids, n_conn, d_arr, i_param);
    // free allocated device memory
    gpuErrchk(hipFree(d_conn_ids));
    gpuErrchk(hipFree(d_arr));
  }
  
  return 0;
}

//////////////////////////////////////////////////////////////////////
// Set the float parameter param_name of an array of n_conn connections,
// identified by the indexes conn_ids[i], to the value val
//////////////////////////////////////////////////////////////////////
int NESTGPU::SetConnectionFloatParam(int64_t *conn_ids, int64_t n_conn,
				     float val,
				     std::string param_name)
{
  // Check if param_name is a connection float parameter
  int i_param = GetConnectionFloatParamIndex(param_name);
  if (i_param < 0) {
    throw ngpu_exception(std::string("Unrecognized connection float parameter ")
			 + param_name);
  }
  if (i_param == i_delay_param) {
        throw ngpu_exception("Connection delay cannot be modified");
  }

  if (n_conn > 0) {
    // declare pointers to arrays in device memory
    int64_t *d_conn_ids;
    // allocate array of connection ids in device memory
    // and copy the ids from host to device array
    gpuErrchk(hipMalloc(&d_conn_ids, n_conn*sizeof(int64_t)));
    gpuErrchk(hipMemcpy(d_conn_ids, conn_ids, n_conn*sizeof(int64_t),
			 hipMemcpyHostToDevice));
        
    // launch kernel to set connection parameters
    SetConnectionFloatParamKernel<<<(n_conn+1023)/1024, 1024 >>>
      (d_conn_ids, n_conn, val, i_param);
    // free allocated device memory
    gpuErrchk(hipFree(d_conn_ids));    
  }
  
  return 0;
}


//////////////////////////////////////////////////////////////////////
// Set the integer parameter param_name of an array of n_conn connections,
// identified by the indexes conn_ids[i], using the values from the array
// h_param_arr
//////////////////////////////////////////////////////////////////////
int NESTGPU::SetConnectionIntParamArr(int64_t *conn_ids, int64_t n_conn,
				      int *h_param_arr,
				      std::string param_name)
{
  // Check if param_name is a connection int parameter
  int i_param = GetConnectionIntParamIndex(param_name);
  if (i_param < 0) {
    throw ngpu_exception(std::string("Unrecognized connection int parameter ")
			 + param_name);
  }
  if (i_param == i_source_param) {
    throw ngpu_exception("Connection source node cannot be modified");
  }

  if (n_conn > 0) {
    // declare pointers to arrays in device memory
    int64_t *d_conn_ids;
    int *d_arr;
    // allocate array of connection ids in device memory
    // and copy the ids from host to device array
    gpuErrchk(hipMalloc(&d_conn_ids, n_conn*sizeof(int64_t)));
    gpuErrchk(hipMemcpy(d_conn_ids, conn_ids, n_conn*sizeof(int64_t),
			 hipMemcpyHostToDevice));
    
    // allocate connection parameter array in device memory
    gpuErrchk(hipMalloc(&d_arr, n_conn*sizeof(int)));

    // copy connection parameter array from host to device memory
    gpuErrchk(hipMemcpy(d_arr, h_param_arr, n_conn*sizeof(int),
			 hipMemcpyHostToDevice));
    
    // launch kernel to set connection parameters
    SetConnectionIntParamKernel<<<(n_conn+1023)/1024, 1024 >>>
      (d_conn_ids, n_conn, d_arr, i_param);
    // free allocated device memory
    gpuErrchk(hipFree(d_conn_ids));
    gpuErrchk(hipFree(d_arr));

  }
  
  return 0;
}

//////////////////////////////////////////////////////////////////////
// Set the int parameter param_name of an array of n_conn connections,
// identified by the indexes conn_ids[i], to the value val
//////////////////////////////////////////////////////////////////////
int NESTGPU::SetConnectionIntParam(int64_t *conn_ids, int64_t n_conn,
				   int val, std::string param_name)
{
  // Check if param_name is a connection float parameter
  int i_param = GetConnectionIntParamIndex(param_name);
  if (i_param < 0) {
    throw ngpu_exception(std::string("Unrecognized connection int parameter ")
			 + param_name);
  }
  if (i_param == i_source_param) {
    throw ngpu_exception("Connection source node cannot be modified");
  }

  if (n_conn > 0) {
    // declare pointers to arrays in device memory
    int64_t *d_conn_ids;
    // allocate array of connection ids in device memory
    // and copy the ids from host to device array
    gpuErrchk(hipMalloc(&d_conn_ids, n_conn*sizeof(int64_t)));
    gpuErrchk(hipMemcpy(d_conn_ids, conn_ids, n_conn*sizeof(int64_t),
			 hipMemcpyHostToDevice));
        
    // launch kernel to set connection parameters
    SetConnectionIntParamKernel<<<(n_conn+1023)/1024, 1024 >>>
      (d_conn_ids, n_conn, val, i_param);
    // free allocated device memory
    gpuErrchk(hipFree(d_conn_ids));
  }
  
  return 0;
}

